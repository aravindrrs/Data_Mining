#include "hip/hip_runtime.h"
 
#include <stdio.h>
#include <fcntl.h>
#include <stdlib.h>
#include <sys/time.h>
#include <sys/stat.h>
#include <unistd.h>
#include <string.h>
#include <math.h>
#include <stdint.h>
#include <hiprand/hiprand_kernel.h>
#include <hiprand.h>
#include "common.h"
#include "VisualCryptographyGPU.h"
 
__constant__ level WhiteShare1_d[2][2] = {0,0,1,1};
__constant__ level WhiteShare2_d[2][2] = {0,0,1,1};
__constant__ level BlackShare1_d[2][2] = {0,0,1,1};
__constant__ level BlackShare2_d[2][2] = {1,1,0,0};

void CheckCUDAError(const char *msg)
{
     hipError_t code =hipGetLastError();
      if(code!=hipSuccess)
      {
          fprintf(stderr,"Cuda Error: %s: %s.\n",msg,hipGetErrorString(code));
          exit(EXIT_FAILURE);
      }
}

__global__ void CodecKernel(level *pImage_d,level *pShare1_d, level *pShare2_d, int iWidth, int iHeight, int iCodecPath)
{

// ********************************************************************************
     										  									  *
// 1) If iCodecPath=ENCODE, the code performs encoding. In that case              *
//    input image of size (iHeight x iWidth) pixels of (type level) stored at GPU *
//    address pImage_d. The share images of size (2*iHeight x 2*iWidth) are stored*
//    at GPU addresses pShare1_d and pShare2_d.                                   *
//            																	  *
// 2) If iCodecPath=DECODE, the code performs decoding. In that case              *
//    output image of size (iHeight x iWidth) pixels of (type level) is stored    *
//    at GPU address pImage_d. The share images of size (iHeight x iWidth) are    *
//    stored at GPU addresses pShare1_d and pShare2_d.                            *
// ********************************************************************************
int randomval;
hiprandState_t state;


int x=threadIdx.x+blockIdx.x*blockDim.x;
int y = blockIdx.y * blockDim.y + threadIdx.y;	    


if(iCodecPath==0)
{
if(x<iWidth && y<iHeight)
{
hiprand_init(y*x,1,0,&state);
if(*(pImage_d+x+iWidth*y)==BLACK)
{
randomval=hiprand(&state)%2;
		 if(randomval==0)   
          {
		    
        	*(pShare1_d+2*y*iWidth*2 +2*x)=0;
            *(pShare1_d+2*y*iWidth*2 +2*x+1)=0;
            *(pShare1_d+2*y*iWidth*2 +2*x+2*iWidth)=1;
            *(pShare1_d+2*y*iWidth*2 +2*x+2*iWidth+1)=1;
            } 
			else
			{			 
   		    *(pShare1_d+2*y*iWidth*2 +2*x)=1;
            *(pShare1_d+2*y*iWidth*2 +2*x+1)=1;
            *(pShare1_d+2*y*iWidth*2 +2*x+2*iWidth)=0;
            *(pShare1_d+2*y*iWidth*2 +2*x+2*iWidth+1)=0;
			}		            
            		
			*(pShare2_d+2*y*iWidth*2 +2*x)=1-*(pShare1_d+2*y*iWidth*2 +2*x);
            *(pShare2_d+2*y*iWidth*2 +2*x+1)=1-*(pShare1_d+2*y*iWidth*2 +2*x+1);
            *(pShare2_d+2*y*iWidth*2 +2*x+2*iWidth)=1-*(pShare1_d+2*y*iWidth*2 +2*x+2*iWidth);
            *(pShare2_d+2*y*iWidth*2 +2*x+2*iWidth+1)=1-*(pShare1_d+2*y*iWidth*2 +2*x+2*iWidth+1);
			}
			else
			//else if(*(pImage_d+x+iWidth*y)==WHITE)
			{
			randomval=hiprand(&state)%2;
          		
			 if(randomval==0)   
              {
		    
        	*(pShare1_d+2*y*iWidth*2 +2*x)=0;
            *(pShare1_d+2*y*iWidth*2 +2*x+1)=0;
            *(pShare1_d+2*y*iWidth*2 +2*x+2*iWidth)=1;
            *(pShare1_d+2*y*iWidth*2 +2*x+2*iWidth+1)=1;
            } 
			else
			{			 
   		    *(pShare1_d+2*y*iWidth*2 +2*x)=1;
            *(pShare1_d+2*y*iWidth*2 +2*x+1)=1;
            *(pShare1_d+2*y*iWidth*2 +2*x+2*iWidth)=0;
            *(pShare1_d+2*y*iWidth*2 +2*x+2*iWidth+1)=0;
			} 
			
            *(pShare2_d+2*y*iWidth*2 +2*x)=*(pShare1_d+2*y*iWidth*2 +2*x);
            *(pShare2_d+2*y*iWidth*2 +2*x+1)=*(pShare1_d+2*y*iWidth*2 +2*x+1);
            *(pShare2_d+2*y*iWidth*2 +2*x+2*iWidth)=*(pShare1_d+2*y*iWidth*2 +2*x+2*iWidth);
            *(pShare2_d+2*y*iWidth*2 +2*x+2*iWidth+1)=*(pShare1_d+2*y*iWidth*2 +2*x+2*iWidth+1);
            
			}
			} 
			}
 
 if(iCodecPath==1)
 {
 if(x<iWidth && y<iHeight)
 {
   *(pImage_d+x+iWidth*y)=!(*(pShare1_d+x+iWidth*y) || *(pShare2_d+x+iWidth*y));    
  }
  }
			  

}    


void VCEncoderGPU(ImageData *pcShare1, ImageData *pcShare2, ImageData *pcImageData, TimeRecord *pTR)
{
 	level *pShare1, *pShare2;
	level *pShare1_d, *pShare2_d;
	level *pImage_d;
	struct timeval start,stop;
	int blockSizeX, blockSizeY, gridSizeX, gridSizeY;
	
	printf("GPU Encoding... \n");
	
	//----------Fill in shares----------//	
	pcShare1->iWidth  = 2 * pcImageData->iWidth;
	pcShare1->iHeight = 2 * pcImageData->iHeight;
	pcShare2->iWidth  = 2 * pcImageData->iWidth;
	pcShare2->iHeight = 2 * pcImageData->iHeight;
    hipDeviceReset();
	//Fill in file header
	memcpy(&(pcShare1->cBmpFH), &(pcImageData->cBmpFH), sizeof(BitMapFileHeader));
	memcpy(&(pcShare2->cBmpFH), &(pcImageData->cBmpFH), sizeof(BitMapFileHeader));
	//Fill in info header
	memcpy(&(pcShare1->cBmpIH), &(pcImageData->cBmpIH), sizeof(BitMapInfoHeader));
	memcpy(&(pcShare2->cBmpIH), &(pcImageData->cBmpIH), sizeof(BitMapInfoHeader));
	//Fill in color table
	memcpy(pcShare1->cBmpImage, pcImageData->cBmpImage, 2*sizeof(BitMapImage));
	memcpy(pcShare2->cBmpImage, pcImageData->cBmpImage, 2*sizeof(BitMapImage));

	//----------Adjust shares----------//
	sprintf(pcShare1->imageName, "Share1G_%s",pcImageData->imageName); //adjust name
	sprintf(pcShare2->imageName, "Share2G_%s",pcImageData->imageName);
	pcShare1->cBmpIH.width  = pcShare1->iWidth; //adjust width
	pcShare2->cBmpIH.width  = pcShare2->iWidth;
	pcShare1->cBmpIH.height = pcShare1->iHeight;  //adjust height
	pcShare2->cBmpIH.height = pcShare2->iHeight;  
	pcShare1->cBmpIH.biSizeImage = pcShare1->cBmpIH.height * (((pcShare1->cBmpIH.bitPix * pcShare1->cBmpIH.width + 31) / 32) * 4); //adjust image size
	pcShare2->cBmpIH.biSizeImage = pcShare2->cBmpIH.height * (((pcShare2->cBmpIH.bitPix * pcShare2->cBmpIH.width + 31) / 32) * 4); //adjust image size	
	pcShare1->cBmpFH.bfSize = pcShare1->cBmpIH.biSizeImage + pcShare1->cBmpFH.bfOffBits;
	pcShare2->cBmpFH.bfSize = pcShare2->cBmpIH.biSizeImage + pcShare2->cBmpFH.bfOffBits;

	//------Generate shares pixels---------//
	pcShare1->imgData = (level *)malloc(4 * pcImageData->iHeight * pcImageData->iWidth * sizeof(level));
	pcShare2->imgData = (level *)malloc(4 * pcImageData->iHeight * pcImageData->iWidth * sizeof(level));
	pShare1 = pcShare1->imgData;
	pShare2 = pcShare2->imgData;
	 	
	gettimeofday(&start,0);

	//------GPU Memory Preparation-------//
	hipMalloc( (void**)&pImage_d, pcImageData->iHeight * pcImageData->iWidth * sizeof(level));
	CheckCUDAError("Original Image GPU Memory Allocation Failed");
	hipMalloc( (void**)&pShare1_d, 4 * pcImageData->iHeight * pcImageData->iWidth * sizeof(level));
	CheckCUDAError("Share1 GPU Memory Allocation Failed");
	hipMalloc( (void**)&pShare2_d, 4 * pcImageData->iHeight * pcImageData->iWidth * sizeof(level));
	CheckCUDAError("Share2 GPU Memory Allocation Failed");
	
	//-------Transfer orignal image-------//
	hipMemcpy(pImage_d, pcImageData->imgData, pcImageData->iHeight * pcImageData->iWidth * sizeof(level), hipMemcpyHostToDevice);
	CheckCUDAError("Copy Original Image to GPU Failed");
 
	gettimeofday(&stop,0);
	pTR->MemTransferTime += ((stop.tv_sec - start.tv_sec) * 1000000 + (stop.tv_usec - start.tv_usec)) / 1000;
	
	gettimeofday(&start,0);
	//-----GPU Kernel Launch-----//
 
	blockSizeX =30;
	blockSizeY =30;
	gridSizeX  =872;
	gridSizeY  =274;


	printf("|--Block Config: %d x %d\n",blockSizeX,blockSizeY);
	printf("|--Grid  Config: %d x %d\n",gridSizeX,gridSizeY);	
	dim3 blocksInGrid(gridSizeX,gridSizeY);
	dim3 threadsInBlock(blockSizeX,blockSizeY);
	CodecKernel<<<blocksInGrid, threadsInBlock>>>(pImage_d,pShare1_d,pShare2_d,pcImageData->iWidth, pcImageData->iHeight, ENCODE);
	hipDeviceSynchronize();
	CheckCUDAError("Encryption Kernel Failed");

	gettimeofday(&stop,0);
	pTR->EncryptionTime += ((stop.tv_sec - start.tv_sec) * 1000000 + (stop.tv_usec - start.tv_usec)) / 1000;

	gettimeofday(&start,0);
	
	//------Transfer back shares------//
	hipMemcpy(pShare1,pShare1_d, 4 * pcImageData->iHeight * pcImageData->iWidth * sizeof(level), hipMemcpyDeviceToHost);
	CheckCUDAError("Copy Share1 to CPU Failed");
	hipMemcpy(pShare2,pShare2_d, 4 * pcImageData->iHeight * pcImageData->iWidth * sizeof(level), hipMemcpyDeviceToHost);
	CheckCUDAError("Copy Share2 to CPU Failed");

	gettimeofday(&stop,0);
	pTR->MemTransferTime += ((stop.tv_sec - start.tv_sec) * 1000000 + (stop.tv_usec - start.tv_usec)) / 1000;

	hipFree(pImage_d);
	hipFree(pShare1_d);
	hipFree(pShare2_d);
}

void VCDecoderGPU(ImageData *pcShare1, ImageData *pcShare2, char *pInputImageName, ImageData *pcImageData, TimeRecord *pTR)
{
 	level *pShare1, *pShare2;
	level *pShare1_d, *pShare2_d;
	level *pImage_d;
	struct timeval start,stop;
	int blockSizeX, blockSizeY, gridSizeX, gridSizeY;
	
	printf("GPU Decoding ...\n");
	pShare1 = pcShare1->imgData;
	pShare2 = pcShare2->imgData;
    hipDeviceReset();
     
	//------GPU Memory Preparation-------//	
	memcpy(pcImageData, pcShare1, sizeof(ImageData));
	pcImageData->imgData = (level *)malloc(pcImageData->iHeight * pcImageData->iWidth * sizeof(level));
	sprintf(pcImageData->imageName, "ReconG_%s",pInputImageName);
	//Memory Allocation 
	hipMalloc( (void**)&pImage_d, pcImageData->iHeight * pcImageData->iWidth * sizeof(level));
	CheckCUDAError("Reconstructed Image GPU Memory Allocation Failed");
	hipMalloc( (void**)&pShare1_d, pcImageData->iHeight * pcImageData->iWidth * sizeof(level));
	CheckCUDAError("Share1 GPU Memory Allocation Failed");
	hipMalloc( (void**)&pShare2_d, pcImageData->iHeight * pcImageData->iWidth * sizeof(level));
	CheckCUDAError("Share2 GPU Memory Allocation Failed");
	
	//Transfer shares
	gettimeofday(&start,0);
	hipMemcpy(pShare1_d, pShare1, pcImageData->iHeight * pcImageData->iWidth * sizeof(level), hipMemcpyHostToDevice);
	CheckCUDAError("Copy Share1 to GPU Failed");
	hipMemcpy(pShare2_d, pShare2, pcImageData->iHeight * pcImageData->iWidth * sizeof(level), hipMemcpyHostToDevice);
	CheckCUDAError("Copy Share2 to GPU Failed");
	gettimeofday(&stop,0);
	pTR->MemTransferTimeDecode += ((stop.tv_sec - start.tv_sec) * 1000000 + (stop.tv_usec - start.tv_usec)) / 1000;
	
	//----------GPU Kernel Launch----------//
	gettimeofday(&start,0);

	//Fill in here 
	blockSizeX =30;
	blockSizeY =30;
	gridSizeX  =110;
	gridSizeY  =2500;

	printf("|--Block Config: %d x %d\n",blockSizeX,blockSizeY);
	printf("|--Grid  Config: %d x %d\n",gridSizeX,gridSizeY);	
	dim3 blocksInGrid(gridSizeX,gridSizeY);
	dim3 threadsInBlock(blockSizeX,blockSizeY);
	CodecKernel<<<blocksInGrid, threadsInBlock>>>(pImage_d,pShare1_d,pShare2_d,pcImageData->iWidth, pcImageData->iHeight, DECODE);
	hipDeviceSynchronize();
	CheckCUDAError("Decryption Kernel Failed");
	gettimeofday(&stop,0);
	pTR->DecodeTime += ((stop.tv_sec - start.tv_sec) * 1000000 + (stop.tv_usec - start.tv_usec)) / 1000;
		
	//------Transfer back reconstructed image------//
	gettimeofday(&start,0);
	hipMemcpy(pcImageData->imgData,pImage_d, pcImageData->iHeight * pcImageData->iWidth * sizeof(level), hipMemcpyDeviceToHost);
	CheckCUDAError("Copy Reconstructed image to CPU Failed");
	gettimeofday(&stop,0);
	pTR->MemTransferTimeDecode += ((stop.tv_sec - start.tv_sec) * 1000000 + (stop.tv_usec - start.tv_usec)) / 1000;
	
	//----------Free memory----------//
 	hipFree(pImage_d);
	hipFree(pShare1_d);
	hipFree(pShare2_d);
	
}


 



